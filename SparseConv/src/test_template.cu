#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define LENGTH 128
#define DEBUG 

extern "C" void spmm_conv(void *input_data_t, void *output_data_t, void *kernel_ptr_t, void *kernel_map_t, void *kernel_offset_t, void *kernel_data_t, void *kernel_ptr_sparse_t, void *kernel_map_sparse_t); 

static unsigned CudaTest(const char *msg) {
	
	hipDeviceSynchronize();
	hipError_t e = hipGetLastError();
	if (hipSuccess != (e)) {
		printf("\033[91m");
		printf("%s: %d\n", msg, e); 
		printf("%s in %s at line %d\n", hipGetErrorString(e),__FILE__, __LINE__);
		printf("\033[0m");
		exit(-1);
		//return 1;
	}
	return 0;
}


#define CHECK_KERNELCALL()\
{ \
    const hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) { \
        printf("%s in %s at line %d\n", hipGetErrorString(err),__FILE__, __LINE__); \
        exit(EXIT_FAILURE); \
    } \
}\

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

_CODE_KERNEL

void spmm_conv(void *input_data_t, void *output_data_t, void *kernel_ptr_t, void *kernel_map_t, void *kernel_offset_t, void *kernel_data_t, void *kernel_ptr_sparse_t, void *kernel_map_sparse_t) {
	float *input_data = (float *)input_data_t;
	float *output_data = (float *)output_data_t;
	int *kernel_ptr = (int *)kernel_ptr_t;
	int *kernel_map = (int *)kernel_map_t;
	int *kernel_offset = (int *)kernel_offset_t;
	float *kernel_data = (float *)kernel_data_t;
	int *kernel_ptr_sparse = (int *)kernel_ptr_sparse_t;
	int *kernel_map_sparse = (int *)kernel_map_sparse_t;

	_DECL_STREAM

	_CALL_KERNEL

	CudaTest("Something gone wrong");

	_CLEAN_UP
}


